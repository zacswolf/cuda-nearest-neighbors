#include "hip/hip_runtime.h"
#include "nearest_neighbor_gpu.h"
#include "vector"
#include "math.h"
//#include "cuda_matrix.h"

template <typename T, typename G>
__global__ void gpuNormalKernel(Matrix<T> *d_trainData, Matrix<G> *d_trainLabels, Matrix<T> *d_testData, int *d_closestPoint, double *d_closestDistance, G *d_predictedLabels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // test point index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // train point index

	//printf("i: %d, j: %d\n", i, j);

	int closestPoint = d_closestPoint[i];
	double closestDistance = d_closestDistance[i];
	//printf("before distance: %d\n", (*d_trainData).data[0]);
	double currentDistance = Matrix<T>::l2RowDistanceSeq(*d_trainData, j, *d_testData, i);

	// Save if currentDistance < closestDistance
	if (currentDistance < closestDistance) {
		__syncthreads();
		d_closestPoint[i] = j;
		d_closestDistance[i] = currentDistance;
		//printf("distance: %f\n", currentDistance);
		//printf("closest point: %f\n", d_trainLabels->data[5]);
		d_predictedLabels[i] = d_trainLabels->data[j];
		__syncthreads();
		//printf("d_predictlabels[%d] = %f\n", d_trainLabels->data[closestPoint]);
		//printf("d_predictlabels[%d] = %d\n", i, d_trainLabels->data[closestPoint]);
	}
}

template <typename T, typename G>
G* gpuNormal(Matrix<T> &trainData, Matrix<G> &trainLabels, Matrix<T> &testData) {
	int numPredictPoints = testData.numRows;
	int numDataPoints = trainData.numRows;

	G *predictedLabels = new G(numPredictPoints);
	G *d_predictedLabels;
	hipMalloc(&d_predictedLabels, numPredictPoints * sizeof(G));

	//Matrix<T>(arr, numPoints, dim);
	Matrix<T> *d_trainData, *d_testData;
	int trainDataBytes = (trainData.numRows * trainData.numCols) * sizeof(T);
	int testDataBytes = (testData.numRows * testData.numCols) * sizeof(T);
	//printf("Train data bytes: %d\n", trainDataBytes);
	hipMalloc(&d_trainData, sizeof(Matrix<T>));
	hipMalloc(&d_testData, sizeof(Matrix<T>));
	hipMemcpy(d_trainData, &trainData, sizeof(Matrix<T>), hipMemcpyHostToDevice);
	hipMemcpy(d_testData, &testData, sizeof(Matrix<T>), hipMemcpyHostToDevice);
	// Copy over data as well
	T *trainDataRaw, *testDataRaw;
	hipMalloc(&trainDataRaw, trainDataBytes);
	hipMalloc(&testDataRaw, testDataBytes);
	hipMemcpy(trainDataRaw, trainData.data, trainDataBytes, hipMemcpyHostToDevice);
	hipMemcpy(testDataRaw, testData.data, testDataBytes, hipMemcpyHostToDevice);
	// Set device data pointers
	hipMemcpy((void *)&(d_trainData->data), &trainDataRaw, sizeof(T *), hipMemcpyHostToDevice);
	hipMemcpy((void *)&(d_testData->data), &testDataRaw, sizeof(T *), hipMemcpyHostToDevice);

	Matrix<G> *d_trainLabels;
	int trainLabelsBytes = (trainLabels.numRows * trainLabels.numCols) * sizeof(G);
	hipMalloc(&d_trainLabels, sizeof(Matrix<G>));
	hipMemcpy(d_trainLabels, &trainLabels, sizeof(Matrix<G>), hipMemcpyHostToDevice);
	G *trainLabelsRaw;
	hipMalloc(&trainLabelsRaw, trainLabelsBytes);
	hipMemcpy(trainLabelsRaw, trainLabels.data, trainLabelsBytes, hipMemcpyHostToDevice);
	hipMemcpy((void *)&(d_trainLabels->data), &trainLabelsRaw, sizeof(G *), hipMemcpyHostToDevice);

	int *closestPoint = (int *)calloc(numPredictPoints, sizeof(int));
	int *d_closestPoint;
	hipMalloc(&d_closestPoint, numPredictPoints * sizeof(int));
	double *closestDistance = (double *)malloc(numPredictPoints * sizeof(double));
	for (int i = 0; i < numPredictPoints; i++) { closestDistance[i] = std::numeric_limits<double>::max(); }
	double *d_closestDistance;
	hipMalloc(&d_closestDistance, numPredictPoints * sizeof(double));

	hipMemcpy(d_closestPoint, closestPoint, numPredictPoints * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_closestDistance, closestDistance, numPredictPoints * sizeof(double), hipMemcpyHostToDevice);

	printf("Running on GPU with %d predict points and %d data points \n", numPredictPoints, numDataPoints);
	int xBlock = (int)ceil(((float)numPredictPoints/512.0f));
	int yBlock = (int)ceil(((float)numDataPoints/512.0f));
	printf("block size should be: %d %d\n", xBlock, yBlock);
	dim3 blockSize(xBlock, yBlock);
	int bx = (numPredictPoints + blockSize.x - 1)/blockSize.x;
	int by = (numDataPoints + blockSize.y - 1)/blockSize.y;
	dim3 gridSize = dim3(bx, by);
	gpuNormalKernel<<<gridSize, blockSize>>>(d_trainData, d_trainLabels, d_testData, d_closestPoint, d_closestDistance, d_predictedLabels);
	hipDeviceSynchronize();

	hipMemcpy(predictedLabels, d_predictedLabels, numPredictPoints * sizeof(G), hipMemcpyDeviceToHost);

	// Nearest neighbors
	/*
	for (int currentTestPoint = 0; currentTestPoint < numPredictPoints; currentTestPoint++) {
		int closestPoint = 0;
		double closestDistance = std::numeric_limits<double>::max();
		for (int currentTrainPoint = 0; currentTrainPoint < numDataPoints; currentTrainPoint++) {
			// l2 distance squared
			double currentDistance = Matrix<T>::l2RowDistanceSeq(trainData, currentTrainPoint, testData, currentTestPoint);

			// Save if currentDistance < closestDistance
			bool newClosest = (currentDistance < closestDistance);

			closestPoint = (!newClosest)*closestPoint + newClosest*currentTrainPoint;
			closestDistance = (!newClosest)*closestDistance + newClosest*currentDistance;
		}
		predictedLabels[currentTestPoint] = trainLabels.data[closestPoint];
	}
	*/

	return predictedLabels;
}
template bool* gpuNormal<float, bool>(Matrix<float>&, Matrix<bool>&, Matrix<float>&);

template <typename T, typename G>
G* gpuJLGaussian(Matrix<T> &trainData, Matrix<G> &trainLabels, Matrix<T> &testData, int newDim) {
	int dim = trainData.numCols;

	// Make a random projection matrix of size dim x newDim
	Matrix<float> rpMat = Matrix<float>(dim, newDim);
	std::normal_distribution<float> distribution(0., 1.);
	rpMat.fill(distribution);

	// newData = trainData x rpMat, numDataPoints by newDim
	Matrix<T> newData = Matrix<T>::matMulSeq(trainData, rpMat);

	// newPredict = testData x rpMat, numDataPoints by newDim
	Matrix<T> newPredict = Matrix<T>::matMulSeq(testData, rpMat);

	return seqNormal(newData, trainLabels, newPredict);
}

template <typename T, typename G>
G* gpuJLBernoulli(Matrix<T> &trainData, Matrix<G> &trainLabels, Matrix<T> &testData, int newDim) {
	int dim = trainData.numCols;

	// Make a random projection matrix of size dim x newDim
	Matrix<bool> rpMat = Matrix<bool>(dim, newDim);
	std::bernoulli_distribution distribution(.5);
	rpMat.fill(distribution);

	// newData = trainData x rpMat, numDataPoints by newDim

	Matrix<decltype(std::declval<T&>() * std::declval<G&>())> newData = Matrix<T>::matMulSeq(trainData, rpMat);

	// newPredict = testData x rpMat, numDataPoints by newDim
	Matrix<T> newPredict = Matrix<T>::matMulSeq(testData, rpMat);

	return seqNormal(newData, trainLabels, newPredict);
}

template <typename T, typename G>
G* gpuJLFast(Matrix<T> &trainData, Matrix<G> &trainLabels, Matrix<T> &testData, int newDim) {
		throw NotImplementedException("SEQUENTIAL::JLFAST");
}
