#include "hip/hip_runtime.h"
#include "matrix.h"

template <typename T>
__host__ void Matrix<T>::fill(std::normal_distribution<float> distribution) {
	std::default_random_engine generator(0);

	for (int i = 0; i < (this->numRows * this->numCols); i++) {
		this->data[i] = distribution(generator);
	}
}

template <typename T>
__host__ void Matrix<T>::fill(std::bernoulli_distribution distribution) {
	std::default_random_engine generator(0);

	for (int i = 0; i < (this->numRows * this->numCols); i++) {
		this->data[i] = distribution(generator);
	}
}

template <typename T>
__host__ void Matrix<T>::fill(T val) {
	if (this->device==0){
		for (int i = 0; i < (this->numRows * this->numCols); i++) {
			this->data[i] = val;
		}
	} else {
		hipMemset(this->data, val, (this->numRows * this->numCols)*sizeof(T));
	}
}

template <typename T>
__host__ __device__ int Matrix<T>::index(int row, int col) {
	return row*(this->numCols) + col;
}

// Move matrix between CPU and device
template <typename T>
__host__ Matrix<T> Matrix<T>::toDevice(int device) {
	if (this->device == 0 && device != 0) {
		// assert(device != this->device);


		int dataBytes = (this->numRows * this->numCols) * sizeof(T);

		T *dataRaw;
		hipMalloc(&dataRaw, dataBytes);
		hipMemcpy(dataRaw, this->data, dataBytes, hipMemcpyHostToDevice);


		Matrix<T> ret = Matrix<T>(dataRaw, this->numRows, this->numCols, device);

		return ret;
	} else if (this->device != 0 && device == 0) {
		// Move back to CPU
		// assert(device != this->device);

		int dataBytes = (this->numRows * this->numCols) * sizeof(T);

		T *dataRaw = new T[this->numRows * this->numCols];
		hipMemcpy(dataRaw, this->data, dataBytes, hipMemcpyDeviceToHost);

		Matrix<T> ret = Matrix<T>(dataRaw, this->numRows, this->numCols, device);
		return ret;
	} else {
		throw NotImplementedException("Matrix<T>::toDevice()");
	}
}

// Removes and returns column from data  
template <typename T>
__host__ pair<Matrix<T>, Matrix<T>> Matrix<T>::popColumn(int columnIndex) {
	if (columnIndex < 0){
		columnIndex = this->numCols + columnIndex;
	}

	float *data = new float[this->numRows * (this->numCols - 1)];
	float *column = new float[this->numRows];

	// Get specific elements from data and store in colummn
	for(int row = 0; row < this->numRows; row++) {
		column[row] = this->data[this->index(row, columnIndex)];
	}

	// Copy this->data minus the popped column to a new data matrix

	// Copy first row up to columnIndex
	auto start = this->data;
	auto end = start + columnIndex;
	auto destination = data;
	copy(start, end, destination);

	for(int row = 1; row < this->numRows-1; row++) {
		// Adjust copy start and end as well as destination locations
		start = end+1;
		end += this->numCols;
		destination += this->numCols - 1;

		// Copy from [row-1, columnIndex+1] to (row, columnIndex)
		copy(start, end, destination);
	}

	// Adjust copy start and end as well as destination locations
	// Set end location to the end of the data matrix
	start = end+1;
	end = this->data + (this->numRows * this->numCols);
	destination += this->numCols - 1;

	// Copy from [last row, columnIndex+1] to (last row, last column)
	copy(start, end, destination);

	// mat.numCols--;
	

	return make_pair(Matrix(column, this->numRows, 1), Matrix(data, this->numRows, this->numCols-1));
}

template <typename T>
template <typename G>
__host__ Matrix<decltype(std::declval<T&>() * std::declval<G&>())> Matrix<T>::matMulSeq(Matrix<T> &left, Matrix<G> &right) {
	int dimLeft = left.numRows;
	int dimCenter = left.numCols;
	int dimRight = right.numCols;
	assert(dimCenter == right.numRows);

	Matrix result = Matrix<decltype(std::declval<T&>() * std::declval<G&>())>(dimLeft, dimRight);
	result.fill(0);

	// Matrix Mult
    for (int i = 0; i < dimLeft; i++) {
        for (int j = 0; j < dimRight; j++) {
            for (int k = 0; k < dimCenter; k++) {
                result.data[result.index(i, j)] += left.data[left.index(i, k)] * right.data[right.index(k, j)];
			}
        }
    }
	
	return result;
}

#define TILE_WIDTH 32

template <typename T, typename G>
__global__ void matMulGPUKernel2DShmem(Matrix<T> left, Matrix<G> right, Matrix<decltype(std::declval<T&>() * std::declval<G&>())> result, int dimLeft, int dimRight, int dimCenter) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ T leftCache[TILE_WIDTH][TILE_WIDTH];
	__shared__ T rightCache[TILE_WIDTH][TILE_WIDTH];

	decltype(std::declval<T&>() * std::declval<G&>()) matmulValue = 0;
	for (int m = 0; m < (TILE_WIDTH + dimLeft - 1)/TILE_WIDTH; m++) {
		leftCache[threadIdx.x][threadIdx.y] = left.data[left.index(i, (m * TILE_WIDTH + threadIdx.y))];
		rightCache[threadIdx.x][threadIdx.y] = right.data[right.index((m * TILE_WIDTH + threadIdx.x), j)];
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; k++) {
			matmulValue += leftCache[threadIdx.x][k] * rightCache[k][threadIdx.y];
		}
	}

	//printf("SHMEM Matmul value: %f\n", matmulValue);
	//printf("dimleft: %d, Block idx: %d\n", dimLeft, blockIdx.x);

	result.data[result.index(i, j)] = matmulValue;
}

template <typename T, typename G>
__global__ void matMulGPUKernel2D(Matrix<T> left, Matrix<G> right, Matrix<decltype(std::declval<T&>() * std::declval<G&>())> result, int dimLeft, int dimRight, int dimCenter) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	decltype(std::declval<T&>() * std::declval<G&>()) matmulValue = 0;
	for (int k = 0; k < dimCenter; k++) {
		matmulValue += left.data[left.index(i, k)] * right.data[right.index(k, j)];
	}

	result.data[result.index(i, j)] = matmulValue;
}

template <typename T>
template <typename G>
__host__ Matrix<decltype(std::declval<T&>() * std::declval<G&>())> Matrix<T>::matMulGPU(Matrix<T> &left, Matrix<G> &right) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int dimLeft = left.numRows;
	int dimCenter = left.numCols;
	int dimRight = right.numCols;
	assert(dimCenter == right.numRows);

	printf("Matmul with %d x %d matrix and %d x %d matrix\n", dimLeft, dimCenter, right.numRows, dimRight);

	assert(left.device == right.device);
	assert(left.device != 0);

	Matrix result = Matrix<decltype(std::declval<T&>() * std::declval<G&>())>(dimLeft, dimRight).toDevice(left.device); // TODO: improve this
	result.fill(0);

	// Launching a 2D kernel
	int xBlock = (int)ceil(((float)dimLeft/512.0f));
	int yBlock = (int)ceil(((float)dimRight/512.0f));
	dim3 blockSize(xBlock, yBlock);
	int bx = (dimLeft + blockSize.x - 1)/blockSize.x;
	int by = (dimRight + blockSize.y - 1)/blockSize.y;
	dim3 gridSize = dim3(bx, by);
	hipEventRecord(start);
	matMulGPUKernel2D<<<gridSize, blockSize>>>(left, right, result, dimLeft, dimRight, dimCenter);
	hipEventRecord(stop);
	/*
	//int blockDim = 32;
	dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
	int xGrid = (int)ceil(((float)dimLeft/(float)TILE_WIDTH));
	int yGrid = (int)ceil(((float)dimRight/(float)TILE_WIDTH));
	dim3 gridSize(xGrid, yGrid);
	hipEventRecord(start);
	matMulGPUKernel2DShmem<<<gridSize, blockSize>>>(left, right, result, dimLeft, dimRight, dimCenter);
	hipEventRecord(stop);
	*/

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU matmul took %f ms\n", milliseconds);

	return result;
}

template <typename T>
__host__ __device__ float Matrix<T>::l2RowDistanceSeq(Matrix &left, int leftRow, Matrix &right, int rightRow) {
	int dim = left.numCols;
	assert(dim == right.numCols);

	float currentDistance = 0.;
	for (int d = 0; d < dim; d++) {
		float term = left.data[left.index(leftRow, d)] - right.data[right.index(rightRow, d)];
		currentDistance += term*term;
	}

	return currentDistance;
}

template <typename T>
__host__ __device__ void Matrix<T>::print() {
	if (this->numCols != 1) {
		printf("[\n");
		for (int row = 0; row < this->numRows; row++) {
			printf("[ ");
			for (int col = 0; col < this->numCols; col++) {
				printf("%s ", std::to_string(this->data[this->index(row, col)]).c_str());
				// cout << this->data[this->index(row, col)] << " ";
			}
			printf("]\n");
		}
		printf("]\n");
	} else {
		printf("[");
		for (int row = 0; row < this->numRows; row++) {
			// cout << this->data[this->index(row, 0)] << " ";
			printf("%s ", std::to_string(this->data[this->index(row, 0)]).c_str());
		}
		printf("]\n");
	}
}

// template class Matrix<float>;
// template class Matrix<bool>;
// template class Matrix<int>;
